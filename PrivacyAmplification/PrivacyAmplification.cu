#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <assert.h>
#include <algorithm>
#include <iterator>
#include <math.h>
#include <zmq.h>
#ifdef _WIN32
#include <windows.h>
#endif
#include <thread>
#include <atomic>
#include <bitset>
#include <future>
#include <iostream>
#include <fstream>
#include <sstream>
#include <chrono>
#include <random>
#include "yaml/Yaml.hpp"
#include "sha3/sha3.h"
#if !defined(__NVCC__)
#include "sha3/sha3.c"
#endif
#include "ThreadPool.h"



#define VKFFT_BACKEND 0
//#define __NVCC__

#if defined(__NVCC__)
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#else
#include "vulkan/vulkan.h"
#include "glslang_c_interface.h"
//#include "half_lib/half.hpp"
#include "vuda/vuda_runtime.hpp"
#include "vkFFT/vkFFT.h"
#include "vkFFT/vkFFT_helper.h"
#endif

#if !defined(NDEBUG)
#define VUDA_STD_LAYER_ENABLED
#define VUDA_DEBUG_ENABLED
#endif

#include "PrivacyAmplification.h"
//#define __NVCC__

using namespace std;

//Little endian only!
//#define TEST


#ifdef __HIPCC__
#define KERNEL_ARG2(grid, block) <<< grid, block >>>
#define KERNEL_ARG3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARG4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARG2(grid, block)
#define KERNEL_ARG3(grid, block, sh_mem)
#define KERNEL_ARG4(grid, block, sh_mem, stream)
#endif

#if defined(__NVCC__)
#ifdef __INTELLISENSE__
hipError_t hipMemcpyToSymbol(HIP_SYMBOL(Complex symbol), const void* src, size_t count);
hipError_t hipMemcpyToSymbol(HIP_SYMBOL(Real symbol), const void* src, size_t count);
int __float2int_rn(float in);
unsigned int atomicAdd(unsigned int* address, unsigned int val);
#define __syncthreads()
#endif
#endif

#ifdef DEBUG
	#ifdef _WIN32
		#define BREAK __debugbreak();
	#else
		#define BREAK __builtin_trap();
	#endif
#else
	#define BREAK
#endif

#define __FILENAME__ (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define assertZeroThreshold(actual, threshold, testCaseNr) \
if (abs(actual) > threshold) { \
	std::cerr << "AssertionError in function " << __func__ << " in " << __FILENAME__ << ":" << __LINE__ << " on test case " << testCaseNr \
			  << ": Expected abs(" << actual << ") < " << threshold << endl; \
	unitTestsFailed = true;  \
	unitTestsFailedLocal = true; \
	BREAK \
}

#define assertEquals(actual, expected, testCaseNr) \
if (actual != expected) { \
	std::cerr << "AssertEqualsError in function " << __func__ << " in " << __FILENAME__ << ":" << __LINE__ << " on test case " << testCaseNr \
			  << ": Expected " << expected << " but it was " << actual << endl; \
	unitTestsFailed = true;  \
	unitTestsFailedLocal = true; \
	BREAK \
}

#define assertTrue(actual) \
if (!(actual)) { \
	std::cerr << "AssertTrueError in function " << __func__ << " in " << __FILE__ << ":" << __LINE__ << endl; \
	BREAK \
	exit(101); \
}

#if defined(__NVCC__)
#define assertGPU(data, data_len, value) \
hipDeviceSynchronize(); \
cudaAssertValue KERNEL_ARG3(max(data_len / 1024, 1), min(data_len, 1024), 0) (data, value); \
{ \
	hipError_t error = hipDeviceSynchronize(); \
	assertTrue(error == hipSuccess); \
}
#else
#define assertGPU(data, data_len, value) \
*assertKernelValue = value; \
*assertKernelReturnValue = 0; \
vuda::launchKernel("SPIRV/assert.spv", "main", 0, max(data_len / 1024, 1), min(data_len, 1024), data, assertKernelValue, assertKernelReturnValue); \
hipStreamSynchronize(0); \
assertTrue(*assertKernelReturnValue == 0);
#endif

#if defined(__NVCC__)
/*Because cudaCalloc doesn't exist in Cuda let's make our own one using hipMalloc and hipMemset*/
#define cudaCalloc(address, size) if (hipMalloc(address, size) == hipSuccess) hipMemset(*address, 0b00000000, size);
#endif

#define VULKAN_ASSERT_VALUE(data, data_len, value) \
hipDeviceSynchronize(); \
cudaAssertValue KERNEL_ARG3(max(data_len/1024, 1), data_len, 0) (data, value); \
{ \
hipError_t error = hipDeviceSynchronize(); \
assertTrue(error == hipSuccess); \
}

#if STOPWATCH == TRUE
chrono::high_resolution_clock::time_point start;
chrono::high_resolution_clock::time_point checkpoint;
chrono::high_resolution_clock::time_point new_checkpoint;

#define STOPWATCH_START \
start = std::chrono::high_resolution_clock::now(); \
checkpoint = start;

#define STOPWATCH_SAVE(VALUE) \
new_checkpoint = std::chrono::high_resolution_clock::now(); \
VALUE = std::chrono::duration_cast<std::chrono::nanoseconds>(new_checkpoint-checkpoint).count(); \
checkpoint = new_checkpoint;

#define STOPWATCH_TOTAL(VALUE) \
VALUE = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::high_resolution_clock::now()-start).count();
#else
#define STOPWATCH_START
#define STOPWATCH_SAVE(VALUE)
#define STOPWATCH_TOTAL(VALUE)
#endif

string address_seed_in;
string address_key_in;
string address_amp_out;
int32_t* reuse_seed_amount_array;

uint32_t vertical_len;
uint32_t horizontal_len;
uint32_t vertical_block;
uint32_t horizontal_block;
uint32_t desired_block;
uint32_t desired_bytes;
uint32_t key_blocks;
uint32_t input_cache_block_size;
uint32_t output_cache_block_size;
uint32_t* recv_key;
uint32_t** toeplitz_seed;
uint32_t** key_start;
uint32_t* key_start_zero_pos;
uint32_t** key_rest;
uint32_t* key_rest_zero_pos;
uint8_t** Output;
uint32_t* assertKernelValue;
uint32_t* assertKernelReturnValue;
uint32_t vudaChunkSize;
#ifdef TEST
bool doTest = true;
uint8_t* testMemoryHost;
#endif
bool do_xor_key_rest = true;
bool do_compress = true;

#if SHOW_DEBUG_OUTPUT == TRUE
Real* OutputFloat;
#endif
atomic<uint32_t> input_cache_read_pos_seed;
atomic<uint32_t> input_cache_read_pos_key;
atomic<uint32_t> input_cache_write_pos_seed;
atomic<uint32_t> input_cache_write_pos_key;
atomic<uint32_t> output_cache_read_pos;
atomic<uint32_t> output_cache_write_pos;
mutex printlock;
float normalisation_float;
atomic<bool> unitTestsFailed(false);
atomic<bool> unitTestBinInt2floatVerifyResultThreadFailed(false);
atomic<bool> unitTestToBinaryArrayVerifyResultThreadFailed(false);
atomic<bool> vkFFT_planned(false);

#if defined(__NVCC__)
__device__ __constant__ Complex c0_dev;
__device__ __constant__ Real h0_dev;
__device__ __constant__ Real h1_reduced_dev;
__device__ __constant__ Real normalisation_float_dev;
__device__ __constant__ uint32_t sample_size_dev;
__device__ __constant__ uint32_t pre_mul_reduction_dev;

__device__ __constant__ uint32_t intTobinMask_dev[32] =
{
	0b10000000000000000000000000000000,
	0b01000000000000000000000000000000,
	0b00100000000000000000000000000000,
	0b00010000000000000000000000000000,
	0b00001000000000000000000000000000,
	0b00000100000000000000000000000000,
	0b00000010000000000000000000000000,
	0b00000001000000000000000000000000,
	0b00000000100000000000000000000000,
	0b00000000010000000000000000000000,
	0b00000000001000000000000000000000,
	0b00000000000100000000000000000000,
	0b00000000000010000000000000000000,
	0b00000000000001000000000000000000,
	0b00000000000000100000000000000000,
	0b00000000000000010000000000000000,
	0b00000000000000001000000000000000,
	0b00000000000000000100000000000000,
	0b00000000000000000010000000000000,
	0b00000000000000000001000000000000,
	0b00000000000000000000100000000000,
	0b00000000000000000000010000000000,
	0b00000000000000000000001000000000,
	0b00000000000000000000000100000000,
	0b00000000000000000000000010000000,
	0b00000000000000000000000001000000,
	0b00000000000000000000000000100000,
	0b00000000000000000000000000010000,
	0b00000000000000000000000000001000,
	0b00000000000000000000000000000100,
	0b00000000000000000000000000000010,
	0b00000000000000000000000000000001
};


__device__ __constant__ uint32_t ToBinaryBitShiftArray_dev[32] =
{
	#if AMPOUT_REVERSE_ENDIAN == TRUE
	7, 6, 5, 4, 3, 2, 1, 0, 15, 14, 13, 12, 11, 10, 9, 8, 23, 22, 21, 20, 19, 18, 17, 16, 31, 30, 29, 28, 27, 26, 25, 24
	#else
	31, 30, 29, 28, 27, 26, 25, 24, 23, 22, 21, 20, 19, 18, 17, 16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1, 0
	#endif
};
#endif


void printStream(ostream& os) {
	ostringstream& ss = dynamic_cast<ostringstream&>(os);
	printlock.lock();
	cout << ss.str() << flush;
	printlock.unlock();
}


void printlnStream(ostream& os) {
	ostringstream& ss = dynamic_cast<ostringstream&>(os);
	printlock.lock();
	cout << ss.str() << endl;
	printlock.unlock();
}


string convertStreamToString(ostream& os) {
	ostringstream& ss = dynamic_cast<ostringstream&>(os);
	return ss.str();
}

#if defined(__NVCC__)
__global__ void cudaAssertValue(uint32_t* data, uint32_t value) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	assert(data[i] == value);
}
#endif

int unitTestCalculateCorrectionFloat() {
	println("Started CalculateCorrectionFloat Unit Test...");
	bool unitTestsFailedLocal = false;
	#if defined(__NVCC__)
	hipStream_t CalculateCorrectionFloatTestStream;
	hipStreamCreate(&CalculateCorrectionFloatTestStream);
	#else
	const int CalculateCorrectionFloatTestStream = 0;
	#endif
	uint32_t* count_one_of_global_seed_test;
	uint32_t* count_one_of_global_key_test;
	float* correction_float_dev_test;
	uint32_t* sample_size_test;
	hipHostMalloc((void**)&count_one_of_global_seed_test, sizeof(uint32_t));
	hipHostMalloc((void**)&count_one_of_global_key_test, sizeof(uint32_t));
	hipHostMalloc((void**)&correction_float_dev_test, sizeof(float));
	hipHostMalloc((void**)&sample_size_test, sizeof(uint32_t));
	*sample_size_test = pow(2, 6);
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(sample_size_dev), sample_size_test, sizeof(uint32_t));
	#endif
	for (uint32_t i = 0; i < *sample_size_test; ++i) {
		for (uint32_t j = 0; j < *sample_size_test; ++j) {
			*count_one_of_global_seed_test = i;
			*count_one_of_global_key_test = j;
			#if defined(__NVCC__)
			calculateCorrectionFloat KERNEL_ARG4(1, 1, 0, CalculateCorrectionFloatTestStream)(count_one_of_global_seed_test, count_one_of_global_key_test, correction_float_dev_test);
			#else
			vuda::launchKernel("SPIRV/calculateCorrectionFloat.spv", "main", CalculateCorrectionFloatTestStream, 1, 1, count_one_of_global_seed_test, count_one_of_global_key_test, correction_float_dev_test, sample_size_test);
			#endif
			hipStreamSynchronize(CalculateCorrectionFloatTestStream);
			uint64_t cpu_count_multiplied = *count_one_of_global_seed_test * *count_one_of_global_key_test;
			double cpu_count_multiplied_normalized = cpu_count_multiplied / (double)*sample_size_test;
			double count_multiplied_normalized_modulo = fmod(cpu_count_multiplied_normalized, 2.0);
			assertZeroThreshold(*correction_float_dev_test - count_multiplied_normalized_modulo, 0.0001, i * *sample_size_test + j);
		}
	}
	*sample_size_test = pow(2, 27);
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(sample_size_dev), sample_size_test, sizeof(uint32_t));
	#endif
	std::mt19937_64 gen(777);
	std::uniform_int_distribution<uint32_t> distrib(pow(2, 25), pow(2, 27));
	for (uint32_t n = 0; n < 4096; ++n) {
		*count_one_of_global_seed_test = distrib(gen);
		*count_one_of_global_key_test = distrib(gen);
		#if defined(__NVCC__)
		calculateCorrectionFloat KERNEL_ARG4(1, 1, 0, CalculateCorrectionFloatTestStream)(count_one_of_global_seed_test, count_one_of_global_key_test, correction_float_dev_test);
		#else
		vuda::launchKernel("SPIRV/calculateCorrectionFloat.spv", "main", CalculateCorrectionFloatTestStream, 1, 1, count_one_of_global_seed_test, count_one_of_global_key_test, correction_float_dev_test, sample_size_test);
		#endif
		hipStreamSynchronize(CalculateCorrectionFloatTestStream);
		uint64_t cpu_count_multiplied = *count_one_of_global_seed_test * *count_one_of_global_key_test;
		double cpu_count_multiplied_normalized = cpu_count_multiplied / (double)*sample_size_test;
		double count_multiplied_normalized_modulo = fmod(cpu_count_multiplied_normalized, 2.0);
		assertZeroThreshold(*correction_float_dev_test - count_multiplied_normalized_modulo, 0.0001, n);
	}
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(sample_size_dev), &sample_size, sizeof(uint32_t));
	#endif
	println("Completed CalculateCorrectionFloat Unit Test");
	return unitTestsFailedLocal ? 100 : 0;
}

#if defined(__NVCC__)
__global__
void calculateCorrectionFloat(uint32_t* count_one_of_global_seed, uint32_t* count_one_of_global_key, float* correction_float_dev)
{
	uint64_t count_multiplied = *count_one_of_global_seed * *count_one_of_global_key;
	double count_multiplied_normalized = count_multiplied / (double)sample_size_dev;
	double two = 2.0;
	Real count_multiplied_normalized_modulo = (float)fmod(count_multiplied_normalized, two);
	*correction_float_dev = count_multiplied_normalized_modulo;
}
#endif


#if defined(__NVCC__)
int unitTestSetFirstElementToZero() {
	println("Started SetFirstElementToZero Unit Test...");
	bool unitTestsFailedLocal = false;
	#if defined(__NVCC__)
	hipStream_t SetFirstElementToZeroStreamTest;
	hipStreamCreate(&SetFirstElementToZeroStreamTest);
	#else
	const int SetFirstElementToZeroStreamTest = 0;
	#endif
	float* do1_test;
	float* do2_test;
	hipHostMalloc((void**)&do1_test, pow(2, 10) * 2 * sizeof(float));
	hipHostMalloc((void**)&do2_test, pow(2, 10) * 2 * sizeof(float));
	for (int i = 0; i < pow(2, 10) * 2; ++i) {
		do1_test[i] = i + 0.77;
		do2_test[i] = i + 0.88;
	}
	setFirstElementToZero KERNEL_ARG4(1, 2, 0, SetFirstElementToZeroStreamTest)(reinterpret_cast<Complex*>(do1_test), reinterpret_cast<Complex*>(do2_test));
	hipStreamSynchronize(SetFirstElementToZeroStreamTest);
	assertZeroThreshold(do1_test[0], 0.00001, 0);
	assertZeroThreshold(do1_test[1], 0.00001, 1);
	assertZeroThreshold(do2_test[0], 0.00001, 2);
	assertZeroThreshold(do2_test[1], 0.00001, 3);
	for (int i = 2; i < pow(2, 10) * 2; ++i) {
		assertZeroThreshold(do1_test[i] - (i + 0.77), 0.0001, i * 2);
		assertZeroThreshold(do2_test[i] - (i + 0.88), 0.0001, i * 2 + 1);
	}
	println("Completed SetFirstElementToZero Unit Test");
	return unitTestsFailedLocal ? 100 : 0;
}

__global__
void setFirstElementToZero(Complex* do1, Complex* do2)
{
	if (threadIdx.x == 0) {
		do1[0] = c0_dev;
	}
	else
	{
		do2[0] = c0_dev;
	}
}
#endif


int unitTestElementWiseProduct() {
	println("Started ElementWiseProduct Unit Test...");
	bool unitTestsFailedLocal = false;
	#if defined(__NVCC__)
	hipStream_t ElementWiseProductStreamTest;
	hipStreamCreate(&ElementWiseProductStreamTest);
	#else
	const int ElementWiseProductStreamTest = 0;
	#endif
	uint32_t r = pow(2, 5);
	float* do1_test;
	float* do2_test;
	uint32_t* pre_mul_reduction_test;
	hipHostMalloc((void**)&pre_mul_reduction_test, sizeof(uint32_t));
	*pre_mul_reduction_test = r;
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(pre_mul_reduction_dev), pre_mul_reduction_test, sizeof(uint32_t));
	#endif
	hipHostMalloc((void**)&do1_test, pow(2, 10) * 2 * sizeof(float));
	hipHostMalloc((void**)&do2_test, pow(2, 10) * 2 * sizeof(float));
	for (int i = 0; i < pow(2, 10) * 2; ++i) {
		do1_test[i] = i + 0.77;
		do2_test[i] = i + 0.88;
	}
	#if defined(__NVCC__)
	ElementWiseProduct KERNEL_ARG4((int)((pow(2, 10) + 1023) / 1024), min((int)pow(2, 10), 1024), 0, ElementWiseProductStreamTest)(reinterpret_cast<Complex*>(do1_test), reinterpret_cast<Complex*>(do2_test));
	#else
	vuda::launchKernel("SPIRV/elementWiseProduct.spv", "main", ElementWiseProductStreamTest, (int)((pow(2, 10) + 1023) / 1024), min((int)pow(2, 10), 1024), do1_test, do2_test, pre_mul_reduction_test);
	#endif
	hipStreamSynchronize(ElementWiseProductStreamTest);
	for (int i = 0; i < pow(2, 10) * 2; i += 2) {
		float real = ((i + 0.77) / r) * ((i + 0.88) / r) - (((i + 1) + 0.77) / r) * (((i + 1) + 0.88) / r);
		float imag = ((i + 0.77) / r) * (((i + 1) + 0.88) / r) + (((i + 1) + 0.77) / r) * ((i + 0.88) / r);
		assertZeroThreshold(do1_test[i] - real, 0.001, i);
		assertZeroThreshold(do1_test[i + 1] - imag, 0.001, i + 1);
	}
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(pre_mul_reduction_dev), &pre_mul_reduction, sizeof(uint32_t));
	#endif
	println("Completed ElementWiseProduct Unit Test");
	return unitTestsFailedLocal ? 100 : 0;
}

#if defined(__NVCC__)
__global__
void ElementWiseProduct(Complex* do1, Complex* do2)
{
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	Real r = pre_mul_reduction_dev;
	Complex do1_local = do1[i];
	Complex do2_local = do2[i];
	Real do1x = do1_local.x / r;
	Real do1y = do1_local.y / r;
	Real do2x = do2_local.x / r;
	Real do2y = do2_local.y / r;
	do1_local.x = do1x * do2x - do1y * do2y;
	do1_local.y = do1x * do2y + do1y * do2x;
	do1[i] = do1_local;
}
#endif


//David W. Wilson: https://oeis.org/A000788/a000788.txt
unsigned A000788(unsigned n)
{
	unsigned v = 0;
	for (unsigned bit = 1; bit <= n; bit <<= 1)
		v += ((n >> 1) & ~(bit - 1)) + ((n & bit) ? (n & ((bit << 1) - 1)) - (bit - 1) : 0);
	return v;
}

void unitTestBinInt2floatVerifyResultThread(float* floatOutTest, int i, int i_max)
{
	bool unitTestsFailedLocal = false;
	const Real float0 = 0.0f;
	const Real float1_reduced = 1.0f / reduction;
	for (; i < i_max; ++i) {
		if (((i / 32) & (1 << (31 - (i % 32)))) == 0) {
			assertEquals(floatOutTest[i], float0, i)
		}
		else
		{
			assertEquals(floatOutTest[i], float1_reduced, i)
		}
	}
	if (unitTestsFailedLocal) {
		unitTestBinInt2floatVerifyResultThreadFailed = true;
	}
}

int unitTestBinInt2float() {
	println("Started TestBinInt2float Unit Test...");
	atomic<bool> unitTestsFailedLocal(false);
	#if defined(__NVCC__)
	hipStream_t BinInt2floatStreamTest;
	hipStreamCreate(&BinInt2floatStreamTest);
	#else
	const int BinInt2floatStreamTest = 0;
	float* float1_reduced_test_dev;
	hipHostMalloc((void**)&float1_reduced_test_dev, sizeof(float));
	float float1_reduced_test = 1.0f / reduction;
	*float1_reduced_test_dev = float1_reduced_test;
	#endif
	uint32_t* binInTest;
	float* floatOutTest;
	hipHostMalloc((void**)&binInTest, (pow(2, 27) / 32) * sizeof(uint32_t));
	hipHostMalloc((void**)&floatOutTest, pow(2, 27) * sizeof(float));
	uint32_t* count_one_test;
	hipHostMalloc((void**)&count_one_test, sizeof(uint32_t));

	const auto processor_count = std::thread::hardware_concurrency();
	for (int i = 0; i < pow(2, 27) / 32; ++i) {
		binInTest[i] = i;
	}
	unitTestBinInt2floatVerifyResultThreadFailed = false;
	#if defined(__NVCC__)
	for (uint32_t sample_size_test_exponent = 10; sample_size_test_exponent <= 27; ++sample_size_test_exponent)
	#else
	for (uint32_t sample_size_test_exponent = 10; sample_size_test_exponent <= 26; ++sample_size_test_exponent)
	#endif
	{
		int elementsToCheck = pow(2, sample_size_test_exponent);
		println("TestBinInt2float Unit Test with 2^" << sample_size_test_exponent << " samples...");
		uint32_t sample_size_test = elementsToCheck;
		uint32_t count_one_expected = A000788((sample_size_test / 32) - 1);
		*count_one_test = 0;
		memset(floatOutTest, 0xFF, pow(2, 27) * sizeof(float));
		#if defined(__NVCC__)
		binInt2float KERNEL_ARG4((int)(((int)(sample_size_test)+1023) / 1024), min(sample_size_test, 1024), 0, BinInt2floatStreamTest) (binInTest, floatOutTest, count_one_test);
		#else
		vuda::launchKernel("SPIRV/binInt2float.spv", "main", BinInt2floatStreamTest, (int)(((int)(sample_size_test)+1023) / 1024), min(sample_size_test, 1024), binInTest, floatOutTest, count_one_test, float1_reduced_test_dev);
		#endif
		hipStreamSynchronize(BinInt2floatStreamTest);
		assertEquals(*count_one_test, count_one_expected, -1);
		int requiredTotalTasks = elementsToCheck % 1000000 == 0 ? elementsToCheck / 1000000 : (elementsToCheck / 1000000) + 1;
		ThreadPool* unitTestBinInt2floatVerifyResultPool = new ThreadPool(min(max(processor_count, 1), requiredTotalTasks));
		for (int i = 0; i < elementsToCheck; i += 1000000) {
			unitTestBinInt2floatVerifyResultPool->enqueue(unitTestBinInt2floatVerifyResultThread, floatOutTest, i, min(i + 1000000, elementsToCheck));
		}
		unitTestBinInt2floatVerifyResultPool->~ThreadPool();
	}
	if (unitTestBinInt2floatVerifyResultThreadFailed) {
		unitTestsFailedLocal = true;
	}
	println("Completed TestBinInt2float Unit Test");
	return unitTestsFailedLocal ? 100 : 0;
}

#if defined(__NVCC__)
__global__
void binInt2float(uint32_t* binIn, Real* realOut, uint32_t* count_one_global)
{
	//Multicast
	Real h0_local = h0_dev;
	Real h1_reduced_local = h1_reduced_dev;
	__shared__ uint32_t binInShared[32];

	uint32_t block = blockIdx.x;
	uint32_t idx = threadIdx.x;
	uint32_t maskToUse;
	uint32_t inPos;
	uint32_t outPos;
	maskToUse = idx % 32;
	inPos = idx / 32;
	outPos = 1024 * block + idx;

	if (threadIdx.x < 32) {
		binInShared[idx] = binIn[32 * block + idx];
	}
	__syncthreads();

	if ((binInShared[inPos] & intTobinMask_dev[maskToUse]) == 0) {
		realOut[outPos] = h0_local;
	}
	else
	{
		atomicAdd(count_one_global, 1);
		realOut[outPos] = h1_reduced_local;
	}
}
#endif


void unitTestToBinaryArrayVerifyResultThread(uint32_t* binOutTest, uint32_t* key_rest_test, int i, int i_max)
{
	bool unitTestsFailedLocal = false;
	uint32_t mask;
	uint32_t data;
	uint32_t key_rest_little;
	uint32_t key_rest_xor;
	uint32_t actualBit;
	uint32_t expectedBit;
	uint32_t xorBit;
	for (; i < i_max; ++i) {
		mask = 1 << (31 - (i % 32));
		data = binOutTest[i / 32];
		#if AMPOUT_REVERSE_ENDIAN == TRUE
		data = ((((data) & 0xff000000) >> 24) |
			(((data) & 0x00ff0000) >> 8) |
			(((data) & 0x0000ff00) << 8) |
			(((data) & 0x000000ff) << 24));
		#endif
		#if AMPOUT_REVERSE_ENDIAN == TRUE
		key_rest_little = key_rest_test[i / 32];
		key_rest_xor = ((((key_rest_little) & 0xff000000) >> 24) |
			(((key_rest_little) & 0x00ff0000) >> 8) |
			(((key_rest_little) & 0x0000ff00) << 8) |
			(((key_rest_little) & 0x000000ff) << 24));
		#else
		uint32_t key_rest_xor = key_rest_test[i / 32];
		#endif
		actualBit = (data & mask) > 0;
		expectedBit = ((i / 32) & mask) > 0;
		xorBit = (key_rest_xor & mask) > 0;
		expectedBit ^= xorBit;
		assertEquals(actualBit, expectedBit, i)
	}
	if (unitTestsFailedLocal) {
		unitTestToBinaryArrayVerifyResultThreadFailed = true;
	}
}

void unitTestToBinaryArrayVerifyResultThreadNoXOR(uint32_t* binOutTest, int i, int i_max)
{
	bool unitTestsFailedLocal = false;
	uint32_t mask;
	uint32_t data;
	uint32_t actualBit;
	uint32_t expectedBit;
	for (; i < i_max; ++i) {
		mask = 1 << (31 - (i % 32));
		data = binOutTest[i / 32];
#if AMPOUT_REVERSE_ENDIAN == TRUE
		data = ((((data) & 0xff000000) >> 24) |
			(((data) & 0x00ff0000) >> 8) |
			(((data) & 0x0000ff00) << 8) |
			(((data) & 0x000000ff) << 24));
#endif
		actualBit = (data & mask) > 0;
		expectedBit = ((i / 32) & mask) > 0;
		assertEquals(actualBit, expectedBit, i)
	}
	if (unitTestsFailedLocal) {
		unitTestToBinaryArrayVerifyResultThreadFailed = true;
	}
}

int unitTestToBinaryArray() {
	println("Started ToBinaryArray Unit Test...");
	atomic<bool> unitTestsFailedLocal(false);
	#if defined(__NVCC__)
	hipStream_t ToBinaryArrayStreamTest;
	hipStreamCreate(&ToBinaryArrayStreamTest);
	#else
	const int ToBinaryArrayStreamTest = 0;
	#endif
	const Real float0 = 0.0f;
	const Real float1 = 1.0f;
	float* invOutTest;
	uint32_t* binOutTest;
	uint32_t* binOutTestNoXOR;
	uint32_t* key_rest_test;
	Real* correction_float_dev_test;
	hipHostMalloc((void**)&invOutTest, pow(2, 27) * sizeof(float));
	hipHostMalloc((void**)&binOutTest, (pow(2, 27) / 32) * sizeof(uint32_t));
	hipHostMalloc((void**)&binOutTestNoXOR, (pow(2, 27) / 32) * sizeof(uint32_t));
	hipHostMalloc((void**)&key_rest_test, (pow(2, 27) / 32) * sizeof(uint32_t));
	hipHostMalloc((void**)&correction_float_dev_test, sizeof(Real));
	memset(key_rest_test, 0b10101010, (pow(2, 27) / 32) * sizeof(uint32_t));
	*correction_float_dev_test = 1.9f;
	#if defined(__NVCC__)
	float normalisation_float_test = 1.0f;
	hipMemcpyToSymbol(HIP_SYMBOL(normalisation_float_dev), &normalisation_float_test, sizeof(float));
	#else
	float* normalisation_float_test_dev;
	hipHostMalloc((void**)&normalisation_float_test_dev, sizeof(float));
	*normalisation_float_test_dev = 1.0f;
	#endif
	const auto processor_count = std::thread::hardware_concurrency();
	for (int i = 0; i < pow(2, 27); ++i) {
		invOutTest[i] = (((i / 32) & (1 << (31 - (i % 32)))) == 0) ? float0 : float1;
	}
	unitTestToBinaryArrayVerifyResultThreadFailed = false;
	for (uint32_t sample_size_test_exponent = 10; sample_size_test_exponent <= 27; ++sample_size_test_exponent)
	{
		uint32_t sample_size_test = pow(2, sample_size_test_exponent);
		uint32_t vertical_len_test = sample_size_test / 4 + sample_size_test / 8;
		uint32_t elementsToCheck = vertical_len_test;
		uint32_t vertical_block_test = vertical_len_test / 32;
		println("ToBinaryArray Unit Test with 2^" << sample_size_test_exponent << " samples...");
		memset(binOutTest, 0xCC, (pow(2, 27) / 32) * sizeof(uint32_t));
		#if defined(__NVCC__)
		ToBinaryArray KERNEL_ARG4((int)((int)(vertical_block_test) / 31) + 1, 1023, 0, ToBinaryArrayStreamTest) (invOutTest, binOutTest, key_rest_test, correction_float_dev_test);
		ToBinaryArrayNoXOR KERNEL_ARG4((int)((int)(vertical_block_test) / 31) + 1, 1023, 0, ToBinaryArrayStreamTest) (invOutTest, binOutTestNoXOR, correction_float_dev_test);
		#else
		vuda::launchKernel("SPIRV/toBinaryArray.spv", "main", ToBinaryArrayStreamTest, (int)((int)(vertical_block_test) / 31) + 1, 1023, invOutTest, binOutTest, key_rest_test, correction_float_dev_test, normalisation_float_test_dev);
		vuda::launchKernel("SPIRV/toBinaryArrayNoXOR.spv", "main", ToBinaryArrayStreamTest, (int)((int)(vertical_block_test) / 31) + 1, 1023, invOutTest, binOutTestNoXOR, key_rest_test, correction_float_dev_test, normalisation_float_test_dev);
		#endif
		hipStreamSynchronize(ToBinaryArrayStreamTest);
		int requiredTotalTasks = elementsToCheck % 1000000 == 0 ? elementsToCheck / 1000000 : (elementsToCheck / 1000000) + 1;
		ThreadPool* unitTestToBinaryArrayVerifyResultPool = new ThreadPool(min(max(processor_count, 1), requiredTotalTasks));
		for (int i = 0; i < elementsToCheck; i += 1000000) {
			unitTestToBinaryArrayVerifyResultPool->enqueue(unitTestToBinaryArrayVerifyResultThread, binOutTest, key_rest_test, i, min(i + 1000000, elementsToCheck));
			unitTestToBinaryArrayVerifyResultPool->enqueue(unitTestToBinaryArrayVerifyResultThreadNoXOR, binOutTestNoXOR, i, min(i + 1000000, elementsToCheck));
		}
		unitTestToBinaryArrayVerifyResultPool->~ThreadPool();
	}
	if (unitTestToBinaryArrayVerifyResultThreadFailed) {
		unitTestsFailedLocal = true;
	}
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(normalisation_float_dev), &normalisation_float, sizeof(uint32_t));
	#endif
	println("Completed ToBinaryArray Unit Test");
	return unitTestsFailedLocal ? 100 : 0;
}

#if defined(__NVCC__)
__global__
void ToBinaryArray(Real* invOut, uint32_t* binOut, uint32_t* key_rest_local, Real* correction_float_dev)
{
	const Real normalisation_float_local = normalisation_float_dev;
	const uint32_t block = blockIdx.x;
	const uint32_t idx = threadIdx.x;
	const Real correction_float = *correction_float_dev;

	__shared__ uint32_t key_rest_xor[31];
	__shared__ uint32_t binOutRawBit[992];
	if (idx < 992) {
		binOutRawBit[idx] = ((__float2int_rn(invOut[block * 992 + idx] / normalisation_float_local + correction_float) & 1)
			<< ToBinaryBitShiftArray_dev[idx % 32]);
	}
	else if (idx < 1023)
	{
		#if AMPOUT_REVERSE_ENDIAN == TRUE
		uint32_t key_rest_little = key_rest_local[block * 31 + idx - 992];
		key_rest_xor[idx - 992] =
			((((key_rest_little) & 0xff000000) >> 24) |
				(((key_rest_little) & 0x00ff0000) >> 8) |
				(((key_rest_little) & 0x0000ff00) << 8) |
				(((key_rest_little) & 0x000000ff) << 24));
		#else
		key_rest_xor[idx - 992] = key_rest_local[block * 31 + idx - 992];
		#endif
	}
	__syncthreads();

	if (idx < 31) {
		const uint32_t pos = idx * 32;
		uint32_t binOutLocal =
			(binOutRawBit[pos] | binOutRawBit[pos + 1] | binOutRawBit[pos + 2] | binOutRawBit[pos + 3] |
				binOutRawBit[pos + 4] | binOutRawBit[pos + 5] | binOutRawBit[pos + 6] | binOutRawBit[pos + 7] |
				binOutRawBit[pos + 8] | binOutRawBit[pos + 9] | binOutRawBit[pos + 10] | binOutRawBit[pos + 11] |
				binOutRawBit[pos + 12] | binOutRawBit[pos + 13] | binOutRawBit[pos + 14] | binOutRawBit[pos + 15] |
				binOutRawBit[pos + 16] | binOutRawBit[pos + 17] | binOutRawBit[pos + 18] | binOutRawBit[pos + 19] |
				binOutRawBit[pos + 20] | binOutRawBit[pos + 21] | binOutRawBit[pos + 22] | binOutRawBit[pos + 23] |
				binOutRawBit[pos + 24] | binOutRawBit[pos + 25] | binOutRawBit[pos + 26] | binOutRawBit[pos + 27] |
				binOutRawBit[pos + 28] | binOutRawBit[pos + 29] | binOutRawBit[pos + 30] | binOutRawBit[pos + 31])
			^ key_rest_xor[idx];
		binOut[block * 31 + idx] = binOutLocal;
	}
}

__global__
void ToBinaryArrayNoXOR(Real* invOut, uint32_t* binOut, Real* correction_float_dev)
{
	const Real normalisation_float_local = normalisation_float_dev;
	const uint32_t block = blockIdx.x;
	const uint32_t idx = threadIdx.x;
	const Real correction_float = *correction_float_dev;

	__shared__ uint32_t binOutRawBit[992];
	if (idx < 992) {
		binOutRawBit[idx] = ((__float2int_rn(invOut[block * 992 + idx] / normalisation_float_local + correction_float) & 1)
			<< ToBinaryBitShiftArray_dev[idx % 32]);
	}
	__syncthreads();

	if (idx < 31) {
		const uint32_t pos = idx * 32;
		uint32_t binOutLocal =
			(binOutRawBit[pos] | binOutRawBit[pos + 1] | binOutRawBit[pos + 2] | binOutRawBit[pos + 3] |
				binOutRawBit[pos + 4] | binOutRawBit[pos + 5] | binOutRawBit[pos + 6] | binOutRawBit[pos + 7] |
				binOutRawBit[pos + 8] | binOutRawBit[pos + 9] | binOutRawBit[pos + 10] | binOutRawBit[pos + 11] |
				binOutRawBit[pos + 12] | binOutRawBit[pos + 13] | binOutRawBit[pos + 14] | binOutRawBit[pos + 15] |
				binOutRawBit[pos + 16] | binOutRawBit[pos + 17] | binOutRawBit[pos + 18] | binOutRawBit[pos + 19] |
				binOutRawBit[pos + 20] | binOutRawBit[pos + 21] | binOutRawBit[pos + 22] | binOutRawBit[pos + 23] |
				binOutRawBit[pos + 24] | binOutRawBit[pos + 25] | binOutRawBit[pos + 26] | binOutRawBit[pos + 27] |
				binOutRawBit[pos + 28] | binOutRawBit[pos + 29] | binOutRawBit[pos + 30] | binOutRawBit[pos + 31]);
		binOut[block * 31 + idx] = binOutLocal;
	}
}
#endif


void printBin(const uint8_t* position, const uint8_t* end) {
	while (position < end) {
		printf("%s", bitset<8>(*position).to_string().c_str());
		++position;
	}
	cout << endl;
}


void printBin(const uint32_t* position, const uint32_t* end) {
	while (position < end) {
		printf("%s", bitset<32>(*position).to_string().c_str());
		++position;
	}
	cout << endl;
}

void memdump(string const& filename, void* data, size_t const bytes)
{
	fstream myfile = fstream(filename.c_str(), std::ios::out | std::ios::binary);
	myfile.write(reinterpret_cast<char const*>(data), bytes);
	myfile.close();
}

pair<double, double> FletcherFloat(float* data, int count)
{
	double sum1 = 0.0;
	double sum2 = 0.0;
	 
	for (int index = 0; index < count; ++index)
	{
		sum1 += (double)abs(data[index]);
		sum2 += sum1;
	}

	return make_pair(sum1, sum2);
}

bool isFletcherFloat(float* data, int count, const double expectedSum1, const double allowedAbsDeltaSum1, const double expectedSum2, const double allowedAbsDeltaSum2) {
	pair<double, double> result = FletcherFloat(data, count);
	println(std::fixed << std::setprecision(8) << result.first << " | " << result.second);
	return abs(result.first - expectedSum1) < allowedAbsDeltaSum1 && abs(result.second - expectedSum2) < allowedAbsDeltaSum2;
}

inline void key2StartRest() {
	uint32_t* key_start_block = key_start[input_cache_write_pos_key];
	uint32_t* key_rest_block = key_rest[input_cache_write_pos_key];
	uint32_t* key_start_zero_pos_block = key_start_zero_pos + input_cache_write_pos_key;
	uint32_t* key_rest_zero_pos_block = key_rest_zero_pos + input_cache_write_pos_key;

	memcpy(key_start_block, recv_key, horizontal_block * sizeof(uint32_t));
	*(key_start_block + horizontal_block) = *(recv_key + horizontal_block) & 0b10000000000000000000000000000000;

	uint32_t j = horizontal_block;
	for (uint32_t i = 0; i < vertical_block; ++i)
	{
		key_rest_block[i] = ((recv_key[j] << 1) | (recv_key[j + 1] >> 31));
		++j;
	}

	uint32_t new_key_start_zero_pos = horizontal_block + 1;
	if (new_key_start_zero_pos < *key_start_zero_pos_block)
	{
		uint32_t key_start_fill_length = *key_start_zero_pos_block - new_key_start_zero_pos;
		memset(key_start_block + new_key_start_zero_pos, 0b00000000, key_start_fill_length * sizeof(uint32_t));
		*key_start_zero_pos_block = new_key_start_zero_pos;
	}

	uint32_t new_key_rest_zero_pos = desired_block - horizontal_block;
	if (new_key_rest_zero_pos < *key_rest_zero_pos_block)
	{
		uint32_t key_rest_fill_length = *key_rest_zero_pos_block - new_key_rest_zero_pos;
		memset(key_rest_block + new_key_rest_zero_pos, 0b00000000, key_rest_fill_length * sizeof(uint32_t));
		*key_rest_zero_pos_block = new_key_rest_zero_pos;
	}
}


inline void readMatrixSeedFromFile() {
	//Cryptographically random Toeplitz seed generated by XOR a self-generated
	//VeraCrypt key file (PRF: SHA-512) with ANU_20Oct2017_100MB_7
	//from the ANU Quantum Random Numbers Server (https://qrng.anu.edu.au/)
	ifstream seedfile(toeplitz_seed_path, ios::binary);

	if (seedfile.fail())
	{
		cout << "Can't open file \"" << toeplitz_seed_path << "\" => terminating!" << endl;
		exit(103);
		abort();
	}

	seedfile.seekg(0, ios::end);
	size_t seedfile_length = seedfile.tellg();
	seedfile.seekg(0, ios::beg);

	if (seedfile_length < desired_bytes)
	{
		cout << "File \"" << toeplitz_seed_path << "\" is with " << seedfile_length << " bytes too short!" << endl;
		cout << "it is required to be at least " << desired_bytes << " bytes => terminating!" << endl;
		exit(104);
		abort();
	}

	char* toeplitz_seed_char = reinterpret_cast<char*>(toeplitz_seed[input_cache_write_pos_seed]);
	seedfile.read(toeplitz_seed_char, desired_bytes);
	for (uint32_t i = 1; i < input_blocks_to_cache; ++i) {
		memcpy(toeplitz_seed[i], toeplitz_seed[0], input_cache_block_size * sizeof(uint32_t));
	}
}


inline void readKeyFromFile() {
	//Cryptographically random Toeplitz seed generated by XOR a self-generated
	//VeraCrypt key file (PRF: SHA-512) with ANU_20Oct2017_100MB_49
	//from the ANU Quantum Random Numbers Server (https://qrng.anu.edu.au/)
	ifstream keyfile(keyfile_path, ios::binary);

	if (keyfile.fail())
	{
		cout << "Can't open file \"" << keyfile_path << "\" => terminating!" << endl;
		exit(105);
		abort();
	}

	keyfile.seekg(0, ios::end);
	size_t keyfile_length = keyfile.tellg();
	keyfile.seekg(0, ios::beg);

	if (keyfile_length < key_blocks * sizeof(uint32_t))
	{
		cout << "File \"" << keyfile_path << "\" is with " << keyfile_length << " bytes too short!" << endl;
		cout << "it is required to be at least " << key_blocks * sizeof(uint32_t) << " bytes => terminating!" << endl;
		exit(106);
		abort();
	}

	char* recv_key_char = reinterpret_cast<char*>(recv_key);
	keyfile.read(recv_key_char, key_blocks * sizeof(uint32_t));
	key2StartRest();
	for (uint32_t i = 0; i < input_blocks_to_cache; ++i) {
		uint32_t* key_start_zero_pos_block = key_start_zero_pos + i;
		uint32_t* key_rest_zero_pos_block = key_rest_zero_pos + i;
		*key_start_zero_pos_block = *key_start_zero_pos;
		*key_rest_zero_pos_block = *key_rest_zero_pos;
	}
	for (uint32_t i = 1; i < input_blocks_to_cache; ++i) {
		memcpy(key_start[i], key_start[0], input_cache_block_size * sizeof(uint32_t));
		memcpy(key_rest[i], key_rest[0], input_cache_block_size * sizeof(uint32_t));
	}
}


#define ZMQ_RECIVE_DATA_SEED(outbuff, sizeBytes, name) \
if (zmq_recv(socket_seed_in, outbuff, sizeBytes, 0) != sizeBytes) { \
	println("Error receiving " << name << " from Seedserver! Retrying..."); \
	continue; \
}

void reciveDataSeed() {
	void* socket_seed_in = nullptr;
	void* context_seed_in = nullptr;

	if (use_matrix_seed_server)
	{
		context_seed_in = zmq_ctx_new();
		socket_seed_in = zmq_socket(context_seed_in, ZMQ_PULL);
		int hwm = 1;
		zmq_setsockopt(socket_seed_in, ZMQ_RCVHWM, &hwm, sizeof(int));
		zmq_connect(socket_seed_in, address_seed_in.c_str());
	}
	else
	{
		readMatrixSeedFromFile();
	}

	bool recive_toeplitz_matrix_seed = use_matrix_seed_server;
	while (true)
	{
		while (input_cache_write_pos_seed % input_blocks_to_cache == input_cache_read_pos_seed) {
			this_thread::yield();
		}
		if (recive_toeplitz_matrix_seed) {
			ZMQ_RECIVE_DATA_SEED(reuse_seed_amount_array + input_cache_write_pos_seed, sizeof(int32_t), "reuse_seed_amount_array")
			ZMQ_RECIVE_DATA_SEED(toeplitz_seed[input_cache_write_pos_seed], desired_bytes, "data")
			if (show_zeromq_status) {
				println("Seed Block recived");
			}
		}

		#if SHOW_INPUT_DEBUG_OUTPUT == TRUE
		printlock.lock();
		cout << "Toeplitz Seed: ";
		printBin(toeplitz_seed[input_cache_write_pos_seed], toeplitz_seed[input_cache_write_pos_seed] + desired_bytes);
		fflush(stdout);
		printlock.unlock();
		#endif

		input_cache_write_pos_seed = (input_cache_write_pos_seed + 1) % input_blocks_to_cache;
	}

	if (use_matrix_seed_server && recive_toeplitz_matrix_seed) {
		zmq_disconnect(socket_seed_in, address_seed_in.c_str());
		zmq_close(socket_seed_in);
		zmq_ctx_destroy(socket_seed_in);
	}
}

#define ZMQ_RECIVE_DATA_KEY(outbuff, sizeBytes, name) \
if (zmq_recv(socket_key_in, outbuff, sizeBytes, 0) != sizeBytes) { \
	println("Error receiving " << name << " from Keyserver! Retrying..."); \
	goto retry_receiving_key; \
}

void reciveDataKey() {
	void* socket_key_in = nullptr;
	void* context_key_in = nullptr;

	if (use_key_server)
	{
		context_key_in = zmq_ctx_new();
		socket_key_in = zmq_socket(context_key_in, ZMQ_PULL);
		int hwm = 1;
		zmq_setsockopt(socket_key_in, ZMQ_RCVHWM, &hwm, sizeof(int));
		zmq_connect(socket_key_in, address_key_in.c_str());
	}
	else
	{
		readKeyFromFile();
	}

	while (true)
	{
		while (input_cache_write_pos_key % input_blocks_to_cache == input_cache_read_pos_key) {
			this_thread::yield();
		}
		if (use_key_server)
		{
		retry_receiving_key:
			ZMQ_RECIVE_DATA_KEY(&do_xor_key_rest, sizeof(bool), "do_xor_key_rest")
			ZMQ_RECIVE_DATA_KEY(&do_compress, sizeof(bool), "do_compress")
			ZMQ_RECIVE_DATA_KEY(&vertical_block, sizeof(uint32_t), "vertical_blocks")
			vertical_len = vertical_block * 32;
			horizontal_len = sample_size - vertical_len;
			horizontal_block = horizontal_len / 32;
			if (do_xor_key_rest) {
				ZMQ_RECIVE_DATA_KEY(recv_key, key_blocks * sizeof(uint32_t), "data")
				key2StartRest();
			}
			else
			{
				uint32_t* key_start_zero_pos_block = key_start_zero_pos + input_cache_write_pos_key;
				ZMQ_RECIVE_DATA_KEY(key_start[input_cache_write_pos_key], (horizontal_block + 1) * sizeof(uint32_t), "data")
				*(key_start[input_cache_write_pos_key] + horizontal_block) &= 0b10000000000000000000000000000000;
				uint32_t new_key_start_zero_pos = horizontal_block + 1;
				if (new_key_start_zero_pos < *key_start_zero_pos_block)
				{
					uint32_t key_start_fill_length = *key_start_zero_pos_block - new_key_start_zero_pos;
					memset(key_start[input_cache_write_pos_key] + new_key_start_zero_pos, 0b00000000, key_start_fill_length * sizeof(uint32_t));
					*key_start_zero_pos_block = new_key_start_zero_pos;
				}
			}
			if (show_zeromq_status) {
				println("Key Block recived");
			}
		}

		#if SHOW_INPUT_DEBUG_OUTPUT == TRUE
		printlock.lock();
		cout << "Key: ";
		printBin(recv_key, recv_key + key_blocks);
		cout << "Key Start: ";
		printBin(key_start[input_cache_write_pos_key], key_start[input_cache_write_pos_key] + desired_block + 1);
		cout << "Key Rest: ";
		printBin(key_rest[input_cache_write_pos_key], key_rest[input_cache_write_pos_key] + desired_block + 1);
		fflush(stdout);
		printlock.unlock();
		#endif

		input_cache_write_pos_key = (input_cache_write_pos_key + 1) % input_blocks_to_cache;
	}

	if (use_key_server)
	{
		zmq_disconnect(socket_key_in, address_key_in.c_str());
		zmq_close(socket_key_in);
		zmq_ctx_destroy(socket_key_in);
	}
}

string toHexString(const uint8_t* data, uint32_t data_length) {
	std::stringstream ss;
	ss << "{ ";
	for (int i = 0; i < data_length; ++i) {
		ss << std::uppercase << std::hex  << "0x" << std::setw(2) << std::setfill('0') << (int)data[i];
		(i % 8 == 7 && i+1 < data_length)
			? ss << "," << std::endl << "  "
			: ss << ", ";
	}
	ss.seekp(-2, std::ios_base::end);
	ss << " };";
	return ss.str();
}

bool isSha3(const uint8_t* dataToVerify, uint32_t dataToVerify_length, const uint8_t expectedHash[]) {
	sha3_ctx sha3;
	rhash_sha3_256_init(&sha3);
	rhash_sha3_update(&sha3, dataToVerify, dataToVerify_length);
	uint8_t* calculatedHash = (uint8_t*)malloc(32);
	rhash_sha3_final(&sha3, calculatedHash);
	//println(toHexString(calculatedHash, 32));
	return memcmp(calculatedHash, expectedHash, 32) == 0;
}

void verifyData(const uint8_t* dataToVerify) {
	if (isSha3(dataToVerify, vertical_len / 8, ampout_sha3)) {
		println("VERIFIED!");
	}
	else
	{
		println("VERIFICATION FAILED!");
		exit(101);
		abort();
	}
}


void sendData() {
	void* amp_out_socket = nullptr;
	if (host_ampout_server)
	{
		void* amp_out_context = zmq_ctx_new();
		amp_out_socket = zmq_socket(amp_out_context, ZMQ_PUSH);
		int hwm = 1;
		zmq_setsockopt(amp_out_socket, ZMQ_SNDHWM, &hwm, sizeof(int));
		while (zmq_bind(amp_out_socket, address_amp_out.c_str()) != 0) {
			println("Binding to \"" << address_amp_out << "\" failed! Retrying...");
		}
	}

	int32_t ampOutsToStore = store_first_ampouts_in_file;
	fstream ampout_file;
	if (ampOutsToStore != 0) {
		ampout_file = fstream("ampout.bin", ios::out | ios::binary);
	}

	ThreadPool* verifyDataPool = nullptr;
	if (verify_ampout)
	{
		verifyDataPool = new ThreadPool(verify_ampout_threads);
	}
	auto start = chrono::high_resolution_clock::now();
	auto stop = chrono::high_resolution_clock::now();

	while (true) {

		while ((output_cache_read_pos + 1) % output_blocks_to_cache == output_cache_write_pos) {
			this_thread::yield();
		}
		output_cache_read_pos = (output_cache_read_pos + 1) % output_blocks_to_cache;

		if (verify_ampout)
		{
			verifyDataPool->enqueue(verifyData, Output[output_cache_read_pos]);
		}

		if (ampOutsToStore != 0) {
			if (ampOutsToStore > 0) {
				--ampOutsToStore;
			}
			ampout_file.write(const_cast<const char*>(reinterpret_cast<char*>(Output[output_cache_read_pos])), do_compress ? vertical_len / 8 : desired_bytes / 2);
			ampout_file.flush();
			if (ampOutsToStore == 0) {
				ampout_file.close();
			}
		}

		if (host_ampout_server)
		{
		retry_sending_amp_out:
			println("zmq_send: " << (do_compress ? vertical_len / 8 : desired_bytes / 2));
			if (zmq_send(amp_out_socket, Output[output_cache_read_pos], do_compress ? vertical_len / 8 : desired_bytes / 2, 0) != (do_compress ? vertical_len / 8 : desired_bytes / 2)) {
				println("Error sending data to AMPOUT client! Retrying...");
				goto retry_sending_amp_out;
			}
			if (show_zeromq_status) {
				println("Block sent to AMPOUT Client");
			}
		}

		stop = chrono::high_resolution_clock::now();
		auto duration = chrono::duration_cast<chrono::microseconds>(stop - start).count();
		start = chrono::high_resolution_clock::now();

		if (show_ampout >= 0)
		{
			printlock.lock();
			cout << "Blocktime: " << duration / 1000.0 << " ms => " << (1000000.0 / duration) * (sample_size / 1000000.0) << " Mbit/s" << endl;
			if (show_ampout > 0)
			{
				for (size_t i = 0; i < min((do_compress ? vertical_len / 8 : desired_bytes / 2) * sizeof(uint32_t), show_ampout); ++i)
				{
					printf("0x%02X: %s\n", Output[output_cache_read_pos][i], bitset<8>(Output[output_cache_read_pos][i]).to_string().c_str());
				}
			}
			fflush(stdout);
			printlock.unlock();
		}
	}
}


void readConfig() {
	Yaml::Node root;
	cout << "# Reading config.yaml..." << endl;
	try
	{
		Yaml::Parse(root, "config.yaml");
	}
	catch (const Yaml::Exception e)
	{
		cout << "Exception " << e.Type() << ": " << e.what() << endl;
		cout << "Can't open file config.yaml => terminating!" << endl;
		exit(102);
		abort();
	}

	//45555 =>seed_in_alice, 46666 => seed_in_bob
	address_seed_in = root["address_seed_in"].As<string>("tcp://127.0.0.1:45555");
	address_key_in = root["address_key_in"].As<string>("tcp://127.0.0.1:47777");  //key_in
	address_amp_out = root["address_amp_out"].As<string>("tcp://127.0.0.1:48888"); //amp_out

	sample_size = static_cast<int>(round(pow(2, root["factor_exp"].As<uint32_t>(27))));
	reduction = static_cast<int>(round(pow(2, root["reduction_exp"].As<uint32_t>(11))));
	pre_mul_reduction = static_cast<int>(round(pow(2, root["pre_mul_reduction_exp"].As<uint32_t>(5))));
	gpu_device_id_to_use = root["gpu_device_id_to_use"].As<uint32_t>(0);
	input_blocks_to_cache = root["input_blocks_to_cache"].As<uint32_t>(16); //Has to be larger then 1
	output_blocks_to_cache = root["output_blocks_to_cache"].As<uint32_t>(16); //Has to be larger then 1

	reuse_seed_amount = root["reuse_seed_amount"].As<int32_t>(0);
	vertical_len = root["vertical_len"].As<int32_t>(50331648);
	do_xor_key_rest = root["do_xor_key_rest"].As<bool>(true);
	do_compress = root["do_compress"].As<bool>(true);

	show_ampout = root["show_ampout"].As<int32_t>(0);
	show_zeromq_status = root["show_zeromq_status"].As<bool>(true);
	use_matrix_seed_server = root["use_matrix_seed_server"].As<bool>(true);
	use_key_server = root["use_key_server"].As<bool>(true);
	host_ampout_server = root["host_ampout_server"].As<bool>(true);
	store_first_ampouts_in_file = root["store_first_ampouts_in_file"].As<int32_t>(true);

	toeplitz_seed_path = root["toeplitz_seed_path"].As<string>("toeplitz_seed.bin");
	keyfile_path = root["keyfile_path"].As<string>("keyfile.bin");

	verify_ampout = root["verify_ampout"].As<bool>(true);
	verify_ampout_threads = root["verify_ampout_threads"].As<uint32_t>(8);
}


inline void setConsoleDesign() {
	#ifdef _WIN32
	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	CONSOLE_SCREEN_BUFFER_INFO csbi;
	DWORD dwConSize;
	COORD coordScreen = { 0, 0 };
	DWORD cCharsWritten;
	GetConsoleScreenBufferInfo(hConsole, &csbi);
	dwConSize = csbi.dwSize.X * csbi.dwSize.Y;
	FillConsoleOutputAttribute(hConsole,
		FOREGROUND_RED | FOREGROUND_BLUE | FOREGROUND_GREEN | FOREGROUND_INTENSITY | BACKGROUND_BLUE,
		dwConSize, coordScreen, &cCharsWritten);
	SetConsoleTextAttribute(hConsole,
		FOREGROUND_RED | FOREGROUND_BLUE | FOREGROUND_GREEN | FOREGROUND_INTENSITY | BACKGROUND_BLUE);
	#endif
}


#if !defined(__NVCC__)
inline void VkFFTCreateConfiguration(VkGPU* vkGPU, vuda::detail::logical_device* logical_device, float* vkBuffer, VkFFTConfiguration* configuration)
{
	configuration->FFTdim = 1;
	configuration->size[0] = sample_size;
	configuration->size[1] = 1;
	configuration->size[2] = 1;
	configuration->performR2C = true;
	configuration->aimThreads = 1024;
	configuration->registerBoost = true;
	configuration->performHalfBandwidthBoost = true;
	configuration->useLUT = false;
	configuration->normalize = false;
	configuration->device = &vkGPU->device;
	configuration->queue = &vkGPU->queue;
	configuration->fence = &vkGPU->fence;
	//println("offset: " << logical_device->GetBufferDescriptor(vkBuffer).offset);
	configuration->buffer = new VkBuffer{ logical_device->GetBufferDescriptor(vkBuffer).buffer };
	bufferSize = (uint64_t)sizeof(float) * 2 * (sample_size / 2 + 1);
	configuration->bufferSize = &bufferSize;
	configuration->commandPool = &vkGPU->commandPool;
	configuration->physicalDevice = &vkGPU->physicalDevice;
	configuration->isCompilerInitialized = 1;
}


inline void planForwardKeyFFT(VkGPU* vkGPU, vuda::detail::logical_device* logical_device, VkFFTApplication* plan_forward_R2C_key, float* key_buffer)
{
	
	if (vkFFT_planned) {
		deleteVkFFT(plan_forward_R2C_key);
	}
	
	/*Plan of the forward real to complex fast fourier transformation*/
	VkFFTConfiguration plan_forward_R2C_key_configuration = {};
	VkFFTCreateConfiguration(vkGPU, logical_device, key_buffer, &plan_forward_R2C_key_configuration);
	plan_forward_R2C_key_configuration.makeForwardPlanOnly = true;
	plan_forward_R2C_key_configuration.performZeropadding[0] = true;
	if (do_xor_key_rest) {
		plan_forward_R2C_key_configuration.fft_zeropad_left[0] = (plan_forward_R2C_key_configuration.size[0] / 4) + (plan_forward_R2C_key_configuration.size[0] / 16) + 1;
	} else {
		plan_forward_R2C_key_configuration.fft_zeropad_left[0] = horizontal_len + 1;
	}
	plan_forward_R2C_key_configuration.fft_zeropad_right[0] = plan_forward_R2C_key_configuration.size[0];
	VkFFTResult result_forward_FFT_key = initializeVkFFT(plan_forward_R2C_key, plan_forward_R2C_key_configuration);
	if (result_forward_FFT_key != VKFFT_SUCCESS)
	{
		println("Failed to plan FFT key! Error Code: " << result_forward_FFT_key);
		exit(result_forward_FFT_key);
		abort();
	}

	vkFFT_planned = true;
}


inline void planVkFFT(VkGPU* vkGPU, vuda::detail::logical_device* logical_device, VkFFTApplication* plan_forward_R2C_key, VkFFTApplication* plan_forward_R2C_seed, VkFFTApplication* plan_inverse_C2R, float* key_buffer, float* seed_buffer)
{
	if (vkFFT_planned)
	{
		/*Delete CUFFT Plans*/
		deleteVkFFT(plan_forward_R2C_key);
		deleteVkFFT(plan_inverse_C2R);
	}

	planForwardKeyFFT(vkGPU, logical_device, plan_forward_R2C_key, key_buffer);


	/*Plan of the forward real to complex fast fourier transformation*/
	VkFFTConfiguration plan_forward_R2C_seed_configuration = {};
	VkFFTCreateConfiguration(vkGPU, logical_device, seed_buffer, &plan_forward_R2C_seed_configuration);
	VkFFTResult result_forward_FFT_seed = initializeVkFFT(plan_forward_R2C_seed, plan_forward_R2C_seed_configuration);
	plan_forward_R2C_seed_configuration.makeForwardPlanOnly = true;
	if (result_forward_FFT_seed != VKFFT_SUCCESS)
	{
		println("Failed to plan FFT seed! Error Code: " << result_forward_FFT_seed);
		exit(result_forward_FFT_seed);
		abort();
	}

	/*Plan of the inverse real to complex fast fourier transformation*/
	VkFFTConfiguration plan_inverse_C2R_configuration = {};
	VkFFTCreateConfiguration(vkGPU, logical_device, key_buffer, &plan_inverse_C2R_configuration);
	plan_inverse_C2R_configuration.makeInversePlanOnly = true;
	plan_inverse_C2R_configuration.performZeropadding[0] = true;
	plan_inverse_C2R_configuration.fft_zeropad_left[0] = 0;
	plan_inverse_C2R_configuration.fft_zeropad_right[0] = 1;
	plan_inverse_C2R_configuration.frequencyZeroPadding = 1;
	VkFFTResult result_plan_inverse_C2R = initializeVkFFT(plan_inverse_C2R, plan_inverse_C2R_configuration);
	if (result_plan_inverse_C2R != VKFFT_SUCCESS)
	{
		println("Failed to plan IFFT! Error Code: " << result_plan_inverse_C2R);
		exit(result_plan_inverse_C2R);
		abort();
	}

	vkFFT_planned = true;
}
#endif


int main(int argc, char* argv[])
{
	//About
	#if defined(__NVCC__)
	string about = streamToString("# PrivacyAmplificationCuda v" << VERSION << " by Nico Bosshard from " << __DATE__ << " #");
	#else
	string about = streamToString("# PrivacyAmplification v" << VERSION << " by Nico Bosshard from " << __DATE__ << " #");
	#endif
	string border(about.length(), '#');
	cout << border << endl << about << endl << border << endl << endl;

	readConfig();
	for (char** arg = argv; *arg; ++arg) {
		if (strcmp(*arg, "speedtest") == 0) {
			sample_size = pow(2, 27);
			verify_ampout = false;
			use_matrix_seed_server = false;
			use_key_server = false;
			host_ampout_server = false;
		} else if (strcmp(*arg, "--factor_exp") == 0) {
			int factor_exp_arg = atoi(*(arg + 1));
			if (factor_exp_arg < 10) {
				println("--factor_exp needs to be larger then 9")
				return 400;
			}
			sample_size = pow(2, factor_exp_arg);
		}
	}

	horizontal_len = sample_size - vertical_len;
	vertical_block = vertical_len / 32;
	horizontal_block = horizontal_len / 32;
	desired_block = sample_size / 32;
	desired_bytes = sample_size / 8;
	key_blocks = desired_block + 1;
	input_cache_block_size = desired_block;
	output_cache_block_size = (desired_block + 31) * sizeof(uint32_t);
	reuse_seed_amount_array = (int32_t*)calloc(input_blocks_to_cache, sizeof(int32_t));
	recv_key = (uint32_t*)malloc(key_blocks * sizeof(uint32_t));
	key_start_zero_pos = (uint32_t*)malloc(input_blocks_to_cache * sizeof(uint32_t));
	key_rest_zero_pos = (uint32_t*)malloc(input_blocks_to_cache * sizeof(uint32_t));
	vudaChunkSize = min(sample_size * sizeof(float), pow(2, 27)); //Max 128 MB Chunks

	cout << "# PrivacyAmplification with " << sample_size << " bits" << endl << endl;
	setConsoleDesign();

	#if defined(__NVCC__)
	hipSetDevice(gpu_device_id_to_use);
	#else
	hipSetDevice(gpu_device_id_to_use, vudaChunkSize);
	#endif

	#if defined(__NVCC__)
	int driver_version = 0;
	int runtime_version = 0;

	hipDriverGetVersion(&driver_version);
	hipRuntimeGetVersion(&runtime_version);
	println("# CUDA version: " << CUDART_VERSION);
	println("# CUDA Runtime version: " << runtime_version);
	println("# Latest version of CUDA supported by the driver: " << driver_version);

	if (driver_version < runtime_version) {
		println("Your CUDA Version is too new to be supported by your drivers");
		println("Please update your NVidia Drivers on the Host or downgrade CUDA");
		println("Docker: Edit the CUDA image used by the Dockerfile to downgrade CUDA");
		return 707;
	}

	size_t free_memory, total_memory;
	hipMemGetInfo(&free_memory, &total_memory);
	println("# Free Memory: " << free_memory / 1048576 << " MB of " << total_memory / 1048576 << " MB" << endl);

	int devCount;
	hipGetDeviceCount(&devCount);

	printlock.lock();
	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		wcout << "# "  << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
		wcout << "#    Global memory:   " << props.totalGlobalMem / 1048576 << " MB" << endl;
		wcout << "#    Shared memory:   " << props.sharedMemPerBlock / 1024 << " KB" << endl;
		wcout << "#    Constant memory: " << props.totalConstMem / 1024 << " KB" << endl;
		wcout << "#    Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "#    Warp size:         " << props.warpSize << endl;
		wcout << "#    Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "#    Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "#    Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << endl;
	} 
	printlock.unlock();
	println("# Using GPU: " << gpu_device_id_to_use  << endl);
	#endif

	input_cache_read_pos_seed = input_blocks_to_cache - 1;
	input_cache_read_pos_key = input_blocks_to_cache - 1;
	input_cache_write_pos_seed = 0;
	input_cache_write_pos_key = 0;
	output_cache_read_pos = input_blocks_to_cache - 1;
	output_cache_write_pos = 0;

	#if defined(__NVCC__)
	hipStreamCreate(&FFTStream);
	hipStreamCreate(&BinInt2floatKeyStream);
	hipStreamCreate(&BinInt2floatSeedStream);
	hipStreamCreate(&CalculateCorrectionFloatStream);
	hipStreamCreate(&cpu2gpuKeyStartStream);
	hipStreamCreate(&cpu2gpuKeyRestStream);
	hipStreamCreate(&cpu2gpuSeedStream);
	hipStreamCreate(&gpu2cpuStream);
	hipStreamCreate(&SetFirstElementToZeroStream);
	hipStreamCreate(&ElementWiseProductStream);
	hipStreamCreate(&ToBinaryArrayStream);
	#else
	vkGPU.instance = vuda::detail::Instance::GetVkInstance();
	vkGPU.physicalDevice = vuda::detail::Instance::GetPhysicalDevice(gpu_device_id_to_use);
	logical_device = vuda::detail::interface_logical_devices::create(vkGPU.physicalDevice, 0, vudaChunkSize);
	const vuda::detail::thrdcmdpool* thrdcmdpool = logical_device->GetPool(std::this_thread::get_id());
	vkGPU.device = logical_device->GetDeviceHandle();
	vkGPU.commandPool = thrdcmdpool->GetCommandPool();
	vkGPU.queue = logical_device->GetQueue(0);
	vkGPU.fence = thrdcmdpool->GetFence(0);

	#if(VKFFT_BACKEND==0)
	glslang_initialize_process();
	#elif(VKFFT_BACKEND==1)
	hipError_t res = hipSuccess;
	hipError_t res2 = hipSuccess;
	res = hipInit(0);
	if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_INITIALIZE;
	res2 = hipSetDevice(vkGPU->device_id);
	if (res2 != hipSuccess) return VKFFT_ERROR_FAILED_TO_SET_DEVICE_ID;
	res = hipDeviceGet(&vkGPU->device, vkGPU->device_id);
	if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_GET_DEVICE;
	res = hipCtxCreate(&vkGPU->context, 0, vkGPU->device);
	if (res != hipSuccess) return VKFFT_ERROR_FAILED_TO_CREATE_CONTEXT;
	#endif
	#endif

	// Allocate host pinned memory on RAM
	hipHostMalloc((void**)&toeplitz_seed, sizeof(uint32_t*) * input_blocks_to_cache);
	for (int i = 0; i < input_blocks_to_cache; ++i) {
		hipHostMalloc((void**)&toeplitz_seed[i], sizeof(uint32_t) * input_cache_block_size);
	}
	hipHostMalloc((void**)&key_start, sizeof(uint32_t*) * input_blocks_to_cache);
	for (int i = 0; i < input_blocks_to_cache; ++i) {
		hipHostMalloc((void**)&key_start[i], sizeof(uint32_t) * input_cache_block_size);
	}
	hipHostMalloc((void**)&key_rest, sizeof(uint32_t*) * input_blocks_to_cache);
	for (int i = 0; i < input_blocks_to_cache; ++i) {
		hipHostMalloc((void**)&key_rest[i], sizeof(uint32_t) * input_cache_block_size + 31 * sizeof(uint32_t));
	}
	hipHostMalloc((void**)&Output, sizeof(uint8_t*) * output_blocks_to_cache);
	for (int i = 0; i < output_blocks_to_cache; ++i) {
		hipHostMalloc((void**)&Output[i], sizeof(uint8_t) * output_cache_block_size);
	}
	hipHostMalloc((void**)&assertKernelValue, sizeof(uint32_t));
	hipHostMalloc((void**)&assertKernelReturnValue, sizeof(uint32_t));
	hipHostMalloc((void**)&value_dev, sizeof(uint8_t));
#ifdef TEST
	hipHostMalloc((void**)&testMemoryHost, max(sample_size * sizeof(Complex), (sample_size + 992) * sizeof(Real)));
#endif
#if SHOW_DEBUG_OUTPUT == TRUE
	hipHostMalloc((void**)&OutputFloat, sample_size * sizeof(float) * output_blocks_to_cache);
#endif

	//Set key_start_zero_pos and key_rest_zero_pos to their default values
	fill(key_start_zero_pos, key_start_zero_pos + input_blocks_to_cache, desired_block);
	fill(key_rest_zero_pos, key_rest_zero_pos + input_blocks_to_cache, desired_block);

	// Allocate memory on GPU
	hipMalloc((void**)&correction_float_dev, sizeof(float));
	hipMalloc((void**)&count_one_of_global_seed, sizeof(uint32_t));
	hipMalloc((void**)&count_one_of_global_key, sizeof(uint32_t));

	#if defined(__NVCC__)
	cudaCalloc((void**)&di1, (uint64_t)sizeof(float) * 2 * ((sample_size + 992) / 2 + 1));

	/*Toeplitz matrix seed FFT input but this memory region is shared with invOut
	  if toeplitz matrix seed recalculation is disabled for the next block*/
	hipMalloc((void**)&di2, (sample_size + 992) * sizeof(Real));
	#else
	cudaCalloc((void**)&di1, (uint64_t)sizeof(float) * 2 * ((sample_size + 992) / 2 + 1), true);

	/*Toeplitz matrix seed FFT input but this memory region is shared with invOut
	  if toeplitz matrix seed recalculation is disabled for the next block*/
	hipMalloc((void**)&di2, (sample_size + 992) * sizeof(Real), true);
	#endif

	#if defined(__NVCC__)
	/*Key FFT output but this memory region is shared with ElementWiseProduct output as they never conflict*/
	hipMalloc((void**)&do1, sample_size * sizeof(Complex));

	/*Toeplitz Seed FFT output but this memory region is shared with invOut
	  if toeplitz matrix seed recalculation is enabled for the next block (default)*/
	hipMalloc((void**)&do2, max(sample_size * sizeof(Complex), (sample_size + 992) * sizeof(Real)));
	#endif

	
	const Real float0 = 0.0f;
	const Real float1_reduced = 1.0f / reduction;
#if defined(__NVCC__)
	const Complex complex0 = make_float2(0.0f, 0.0f);

	/*Copy constant variables from RAM to GPUs constant memory*/
	hipMemcpyToSymbol(HIP_SYMBOL(c0_dev), &complex0, sizeof(Complex));
	hipMemcpyToSymbol(HIP_SYMBOL(h0_dev), &float0, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(h1_reduced_dev), &float1_reduced, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sample_size_dev), &sample_size, sizeof(uint32_t));
	hipMemcpyToSymbol(HIP_SYMBOL(pre_mul_reduction_dev), &pre_mul_reduction, sizeof(uint32_t));
#else
	hipMalloc((void**)&float1_reduced_dev, sizeof(float));
	hipMemcpy(float1_reduced_dev, &float1_reduced, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&normalisation_float_dev, sizeof(float));
	hipMalloc((void**)&sample_size_dev, sizeof(uint32_t));

	hipMalloc((void**)&pre_mul_reduction_dev, sizeof(uint32_t));
	hipMemcpy(pre_mul_reduction_dev, &pre_mul_reduction, sizeof(uint32_t), hipMemcpyHostToDevice);

	uint32_t* zero_dev;
	hipMalloc((void**)&zero_dev, sizeof(uint32_t));
	hipMemcpy(zero_dev, &zero_cpu, sizeof(uint32_t), hipMemcpyHostToDevice);
#endif

	//return unitTestCalculateCorrectionFloat();
	//#if defined(__NVCC__)
	//return unitTestSetFirstElementToZero();
	//#endif
	//return unitTestElementWiseProduct();
	//return unitTestBinInt2float();
	//return unitTestToBinaryArray();

	for (char** arg = argv; *arg; ++arg) {
		if (strcmp(*arg, "speedtest") == 0) {
			for (int32_t i = 0; i < 2; ++i) {
				switch (i) {
				case 0: reuse_seed_amount = 0; break;
				case 1: reuse_seed_amount = -1; break;
				}
				for (int32_t j = 10; j < 28; ++j) {
					sample_size = pow(2, j);
					vertical_len = sample_size / 4 + sample_size / 8;
					mainloop(true, i, j);
				}
			}
			return 0;
		}
		if (strncmp(*arg, "unitTest", 8) != 0) continue;
		if (strcmp(*arg, "unitTestCalculateCorrectionFloat") == 0) return unitTestCalculateCorrectionFloat();
		#if defined(__NVCC__)
		if (strcmp(*arg, "unitTestSetFirstElementToZero") == 0) return unitTestSetFirstElementToZero();
		#endif
		if (strcmp(*arg, "unitTestElementWiseProduct") == 0) return unitTestElementWiseProduct();
		if (strcmp(*arg, "unitTestBinInt2float") == 0) return unitTestBinInt2float();
		if (strcmp(*arg, "unitTestToBinaryArray") == 0) return unitTestToBinaryArray();
	}

	/*The reciveData function is parallelly executed on a separate thread which we start now*/
	thread threadReciveSeedObj(reciveDataSeed);
	threadReciveSeedObj.detach();
	thread threadReciveKeyObj(reciveDataKey);
	threadReciveKeyObj.detach();

	/*The sendData function is parallelly executed on a separate thread which we start now*/
	thread threadSendObj(sendData);
	threadSendObj.detach();

	mainloop(false, 0, 0);

	// Deallocate memoriey on GPU and RAM
	hipFree(di1);
	hipFree(di2);
	hipFree(invOut);
#if defined(__NVCC__)
	hipFree(do1);
	hipFree(do2);
#endif
	hipFree(Output);
#ifdef TEST
	hipFree(testMemoryHost);
#endif
	return 0;

}
	
//##########################
// Mainloop of main thread #
//##########################
void mainloop(bool speedtest, int32_t speedtest_i, int32_t speedtest_j)
{

	#if STOPWATCH == TRUE
	uint64_t stopwatch_wait_for_input_buffer = 0;
	uint64_t stopwatch_cleaned_memory = 0;
	uint64_t stopwatch_set_count_one_to_zero = 0;
	uint64_t stopwatch_binInt2float_key = 0;
	uint32_t stopwatch_binInt2float_seed = 0;
	uint64_t stopwatch_calculateCorrectionFloat = 0;
	uint64_t stopwatch_fft_key = 0;
	uint64_t stopwatch_fft_seed = 0;
	uint64_t stopwatch_setFirstElementToZero = 0;
	uint64_t stopwatch_elementWiseProduct = 0;
	uint64_t stopwatch_ifft = 0;
	uint64_t stopwatch_wait_for_output_buffer = 0;
	uint64_t stopwatch_toBinaryArray = 0;
	uint64_t stopwatch_total = 0;
	uint64_t stopwatch_total_max = UINT64_MAX;
	#endif

	/*relevant_keyBlocks variables are used to detect dirty memory regions*/
	uint32_t relevant_keyBlocks = horizontal_block + 1;
	uint32_t relevant_keyBlocks_old = 0;
	uint32_t dist_freq = sample_size / 2 + 1;

	#if defined(__NVCC__)
	/*Plan fast fourier transformations*/
	hipfftHandle plan_forward_R2C;
	hipfftHandle plan_inverse_C2R;
	#else
	/*Plan fast fourier transformations*/
	VkFFTApplication plan_forward_R2C_seed = {};
	VkFFTApplication plan_forward_R2C_key = {};
	VkFFTApplication plan_inverse_C2R = {};
	#endif

	horizontal_len = sample_size - vertical_len;
	vertical_block = vertical_len / 32;
	horizontal_block = horizontal_len / 32;
	desired_block = sample_size / 32;
	desired_bytes = sample_size / 8;
	key_blocks = desired_block + 1;
	const uint32_t total_reduction = reduction * pre_mul_reduction;
	normalisation_float = ((float)sample_size) / ((float)total_reduction) / ((float)total_reduction);
	#if defined(__NVCC__)
	hipMemcpyToSymbol(HIP_SYMBOL(normalisation_float_dev), &normalisation_float, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(sample_size_dev), &sample_size, sizeof(uint32_t));

	/*Plan of the forward real to complex fast fourier transformation*/
	hipfftResult result_forward_FFT = hipfftPlan1d(&plan_forward_R2C, sample_size, HIPFFT_R2C, 1);
	if (result_forward_FFT != HIPFFT_SUCCESS)
	{
	println("Failed to plan FFT 1! Error Code: " << result_forward_FFT);
	exit(result_forward_FFT);
	abort();
	}
	
	/* Plan of the inverse complex to real fast fourier transformation */
	hipfftResult result_inverse_FFT = hipfftPlan1d(&plan_inverse_C2R, sample_size, HIPFFT_C2R, 1);
	if (result_inverse_FFT != HIPFFT_SUCCESS)
	{
	println("Failed to plan IFFT 1! Error Code: " << result_inverse_FFT);
	exit(result_inverse_FFT);
	abort();
	}
	#else
	hipMemcpy(normalisation_float_dev, &normalisation_float, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(sample_size_dev, &sample_size, sizeof(uint32_t), hipMemcpyHostToDevice);
	planVkFFT(&vkGPU, logical_device, &plan_forward_R2C_key, &plan_forward_R2C_seed, &plan_inverse_C2R, di1, di2);
	#endif

	int32_t speedtest_nr = -4;
	chrono::high_resolution_clock::time_point speedtest_start;
	chrono::high_resolution_clock::time_point speedtest_stop;

	bool recalculate_seed = true;
	bool is_first_seed = true;

	while(true)
	{		

		/*Spinlock waiting for data provider*/
		if (speedtest) {
			++speedtest_nr;
			if (reuse_seed_amount == 0 || is_first_seed) {
				is_first_seed = false;
				recalculate_seed = true;
			} else {
				recalculate_seed = false;
			}
			speedtest_start = chrono::high_resolution_clock::now();
		} else {
			STOPWATCH_START
				chrono::high_resolution_clock::time_point begin = std::chrono::high_resolution_clock::now();
			if (reuse_seed_amount == 0 || is_first_seed) {
				while ((input_cache_read_pos_seed + 1) % input_blocks_to_cache == input_cache_write_pos_seed) {
					this_thread::yield();
				}
				input_cache_read_pos_seed = (input_cache_read_pos_seed + 1) % input_blocks_to_cache; //Switch read cache
				reuse_seed_amount = reuse_seed_amount_array[input_cache_read_pos_seed];
				is_first_seed = false;
				recalculate_seed = true;
			} else {
				recalculate_seed = false;
			}
			while ((input_cache_read_pos_key + 1) % input_blocks_to_cache == input_cache_write_pos_key) {
				this_thread::yield();
			}
			input_cache_read_pos_key = (input_cache_read_pos_key + 1) % input_blocks_to_cache; //Switch read cache
			STOPWATCH_SAVE(stopwatch_wait_for_input_buffer)
		}

		#if defined(__NVCC__)
		/*Detect dirty memory regions parts*/
		/*Not needed on VkFFT as we can make use of it's native zero padding instead*/
		relevant_keyBlocks_old = relevant_keyBlocks;
		relevant_keyBlocks = horizontal_block + 1;
		if (relevant_keyBlocks_old > relevant_keyBlocks) {
			/*Fill dirty memory regions parts with zeros*/
			hipMemset(di1 + relevant_keyBlocks, 0b00000000, (relevant_keyBlocks_old - relevant_keyBlocks) * sizeof(Real));
		}
		STOPWATCH_SAVE(stopwatch_cleaned_memory)
		#else
		relevant_keyBlocks_old = relevant_keyBlocks;
		relevant_keyBlocks = horizontal_block + 1;
		if (relevant_keyBlocks_old != relevant_keyBlocks) {
			planForwardKeyFFT(&vkGPU, logical_device, &plan_forward_R2C_key, di1);
		}
		#endif
		//hipMemset(di1 + relevant_keyBlocks, 0b00000000, (uint64_t)sizeof(float) * 2 * ((pow(2, 27) + 992) / 2 + 1));
		//hipMemset(di2 + relevant_keyBlocks, 0b00000000, (pow(2, 27) + 992) * sizeof(Real));
		
		if (recalculate_seed) {
			hipMemset(count_one_of_global_seed, 0b00000000, sizeof(uint32_t));
			hipMemset(count_one_of_global_key, 0b00000000, sizeof(uint32_t));
			#ifdef TEST
			if (doTest) {
				assertGPU(count_one_of_global_seed, 1, 0);
				assertGPU(count_one_of_global_key, 1, 0);
				assertTrue(isSha3(reinterpret_cast<uint8_t*>(toeplitz_seed[input_cache_read_pos_seed]), desired_bytes, binInt2float_seed_binIn_hash));
			}
			#endif
			STOPWATCH_SAVE(stopwatch_set_count_one_to_zero)
			#if defined(__NVCC__)
			binInt2float KERNEL_ARG4((int)(((int)(sample_size)+1023) / 1024), min(sample_size, 1024), 0,
				BinInt2floatSeedStream) (toeplitz_seed[input_cache_read_pos_seed], di2, count_one_of_global_seed);
			#else
			vuda::launchKernel("SPIRV/binInt2float.spv", "main", BinInt2floatSeedStream, (int)(((int)(sample_size)+1023) / 1024), min(sample_size, 1024), toeplitz_seed[input_cache_read_pos_seed], di2, count_one_of_global_seed, float1_reduced_dev);
			#endif
			hipStreamSynchronize(BinInt2floatSeedStream);
			#ifdef TEST
			if (doTest) {
				println("binInt2float Seed test started...");
				hipMemcpy(testMemoryHost, di2, sample_size * sizeof(Real), hipMemcpyDeviceToHost);
				assertTrue(isSha3(const_cast<uint8_t*>(testMemoryHost), sample_size * sizeof(Real), binInt2float_seed_floatOut_hash));
				println("binInt2float Seed test completed successfully");
			}
			#endif
			STOPWATCH_SAVE(stopwatch_binInt2float_seed)
		}
		else
		{
			hipMemset(count_one_of_global_key, 0b00000000, sizeof(uint32_t));
			#ifdef TEST
			if (doTest) {
				assertGPU(count_one_of_global_key, 1, 0);
			}
			#endif
			STOPWATCH_SAVE(stopwatch_set_count_one_to_zero)
			#if STOPWATCH == TRUE
			stopwatch_binInt2float_seed = 0;
			#endif
		}
		
		#ifdef TEST
		if (doTest) {
			assertTrue(isSha3(reinterpret_cast<uint8_t*>(key_start[input_cache_read_pos_key]), relevant_keyBlocks * sizeof(uint32_t), binInt2float_key_binIn_hash));
		}
		#endif
		#if defined(__NVCC__)
		binInt2float KERNEL_ARG4((int)((relevant_keyBlocks * 32 + 1023) / 1024), min(relevant_keyBlocks * 32, 1024), 0,
			BinInt2floatKeyStream) (key_start[input_cache_read_pos_key], di1, count_one_of_global_key);
		#else
		vuda::launchKernel("SPIRV/binInt2float.spv", "main", BinInt2floatKeyStream, (int)((relevant_keyBlocks * 32 + 1023) / 1024), min(relevant_keyBlocks * 32, 1024), key_start[input_cache_read_pos_key], di1, count_one_of_global_key, float1_reduced_dev);
		#endif
		hipStreamSynchronize(BinInt2floatKeyStream);
		#ifdef TEST
		if (doTest) {
			println("binInt2float Key test started...");
			hipMemcpy(testMemoryHost, di1, relevant_keyBlocks * 32 * sizeof(Real), hipMemcpyDeviceToHost);
			assertTrue(isSha3(const_cast<uint8_t*>(testMemoryHost), relevant_keyBlocks * 32 * sizeof(Real), binInt2float_key_floatOut_hash));
			println("binInt2float Key test completed successfully");
		}
		#endif
		STOPWATCH_SAVE(stopwatch_binInt2float_key)
		
		#ifdef TEST
		if (doTest) {
			assertGPU(count_one_of_global_key, 1, 41947248);
			assertGPU(count_one_of_global_seed, 1, 67113455);
		}
		#endif
		#if defined(__NVCC__)
		calculateCorrectionFloat KERNEL_ARG4(1, 1, 0, CalculateCorrectionFloatStream)
			(count_one_of_global_key, count_one_of_global_seed, correction_float_dev);
		#else
		vuda::launchKernel("SPIRV/calculateCorrectionFloat.spv", "main", CalculateCorrectionFloatStream, 1, 1, count_one_of_global_key, count_one_of_global_seed, correction_float_dev, sample_size_dev);
		#endif
		hipStreamSynchronize(CalculateCorrectionFloatStream);
		STOPWATCH_SAVE(stopwatch_calculateCorrectionFloat)
		#if defined(__NVCC__)
		hipfftExecR2C(plan_forward_R2C, di1, do1);
		hipDeviceSynchronize();
		STOPWATCH_SAVE(stopwatch_fft_key)
		
		if (recalculate_seed) {
			hipfftExecR2C(plan_forward_R2C, di2, do2);
			hipDeviceSynchronize();
			STOPWATCH_SAVE(stopwatch_fft_seed)
		} else {
			#if STOPWATCH == TRUE
			stopwatch_fft_seed = 0;
			#endif
		}
		Complex* intermediate_key = reinterpret_cast<Complex*>(do1);
		Complex* intermediate_seed = reinterpret_cast<Complex*>(do2);
		invOut = reinterpret_cast<Real*>(di2); //invOut and di2 share together the same memory region
		#else
		vkfftExecR2C(&vkGPU, &plan_forward_R2C_key);
		hipStreamSynchronize(FFTStream);
		STOPWATCH_SAVE(stopwatch_fft_key)
		if (recalculate_seed) {
			vkfftExecR2C(&vkGPU, &plan_forward_R2C_seed);
			hipStreamSynchronize(FFTStream);
			STOPWATCH_SAVE(stopwatch_fft_seed)
		} else {
			#if STOPWATCH == TRUE
			stopwatch_fft_seed = 0;
			#endif
		}
		Complex* intermediate_key = reinterpret_cast<Complex*>(di1);
		Complex* intermediate_seed = reinterpret_cast<Complex*>(di2);
		invOut = reinterpret_cast<Real*>(di1); //invOut and di2 share together the same memory region
		#endif
		#ifdef TEST
		if (doTest) {
			hipMemcpy(testMemoryHost, intermediate_key, 2 * (sample_size / 2 + 1) * sizeof(float), hipMemcpyDeviceToHost);
			for (int i = 0; i < 100; i += 2) {
				println(i << ": " << reinterpret_cast<float*>(testMemoryHost)[i] << "|" << reinterpret_cast<float*>(testMemoryHost)[i + 1]);
			}
			for (int i = sample_size - 50; i < sample_size + 50; i += 2) {
				println(i << ": " << reinterpret_cast<float*>(testMemoryHost)[i] << "|" << reinterpret_cast<float*>(testMemoryHost)[i + 1]);
			}
			assertTrue(isFletcherFloat(reinterpret_cast<float*>(testMemoryHost), 2 * (sample_size / 2 + 1), 169418278.63041568, 200.0, 11374845421549196.0, 20000000000.0));
			hipMemcpy(testMemoryHost, intermediate_seed, 2 * (sample_size / 2 + 1) * sizeof(float), hipMemcpyDeviceToHost);
			assertTrue(isFletcherFloat(reinterpret_cast<float*>(testMemoryHost), 2 * (sample_size / 2 + 1), 214211928.23554835, 200.0, 14378010673396208.0, 20000000000.0));
		}
		#endif
		#if defined(__NVCC__)
		setFirstElementToZero KERNEL_ARG4(1, 2, 0, SetFirstElementToZeroStream) (intermediate_key, intermediate_seed);
		hipStreamSynchronize(SetFirstElementToZeroStream);
		
		#ifdef TEST
		if (doTest) {
			hipMemcpy(testMemoryHost, intermediate_key, 2 * (sample_size / 2 + 1) * sizeof(float), hipMemcpyDeviceToHost);
			for (int i = 0; i < 100; i += 2) {
				println(i << ": " << reinterpret_cast<float*>(testMemoryHost)[i] << "|" << reinterpret_cast<float*>(testMemoryHost)[i + 1]);
			}
			for (int i = sample_size - 50; i < sample_size + 50; i += 2) {
				println(i << ": " << reinterpret_cast<float*>(testMemoryHost)[i] << "|" << reinterpret_cast<float*>(testMemoryHost)[i + 1]);
			}
			assertTrue(isFletcherFloat(reinterpret_cast<float*>(testMemoryHost), 2 * (sample_size / 2 + 1), 169397796.57572800, 200.0, 11372096366664388.0, 20000000000.0));
			hipMemcpy(testMemoryHost, intermediate_seed, 2 * (sample_size / 2 + 1) * sizeof(float), hipMemcpyDeviceToHost);
			assertTrue(isFletcherFloat(reinterpret_cast<float*>(testMemoryHost), 2 * (sample_size / 2 + 1), 214179157.99336109, 200.0, 14373612325878530.0, 20000000000.0));
		}
		#endif
		STOPWATCH_SAVE(stopwatch_setFirstElementToZero)
		ElementWiseProduct KERNEL_ARG4((int)((dist_freq + 1023) / 1024), min((int)dist_freq, 1024), 0, ElementWiseProductStream) (intermediate_key, intermediate_seed);
		#else
		vuda::launchKernel("SPIRV/elementWiseProduct.spv", "main", ElementWiseProductStream, (int)((dist_freq + 1023) / 1024), min((int)dist_freq, 1024), intermediate_key, intermediate_seed, pre_mul_reduction_dev);
		#endif
		hipStreamSynchronize(ElementWiseProductStream);
		#if defined(__NVCC__)
		#ifdef TEST
		if (doTest) {
			hipMemcpy(testMemoryHost, intermediate_key, 2 * (sample_size / 2 + 1) * sizeof(float), hipMemcpyDeviceToHost);
			assertTrue(isFletcherFloat(reinterpret_cast<float*>(testMemoryHost), 2 * (sample_size / 2 + 1) * 2, 414613.13602233, 0.5, 83481560389295.703125, 200000000.0));
		}
		#endif
		#endif
		STOPWATCH_SAVE(stopwatch_elementWiseProduct)
		#if defined(__NVCC__)
		hipfftExecC2R(plan_inverse_C2R, intermediate_key, invOut);
		hipDeviceSynchronize();
		#else
		vkfftExecC2R(&vkGPU, &plan_inverse_C2R);
		hipStreamSynchronize(FFTStream);
		#endif
		STOPWATCH_SAVE(stopwatch_ifft)


		/*Spinlock waiting for the data consumer*/
		if (!speedtest) {
			while (output_cache_write_pos % output_blocks_to_cache == output_cache_read_pos) {
				this_thread::yield();
			}
			STOPWATCH_SAVE(stopwatch_wait_for_output_buffer)
		}

		/*Calculates where in the host pinned output memory the Privacy Amplification result will be stored*/
		#ifdef TEST
		if (doTest) {
			hipMemcpy(testMemoryHost, invOut, sample_size * sizeof(Real), hipMemcpyDeviceToHost);
			for (int i = 0; i < 100; i += 2) {
				println(i << ": " << reinterpret_cast<float*>(testMemoryHost)[i] << "|" << reinterpret_cast<float*>(testMemoryHost)[i + 1]);
			}
			for (int i = sample_size - 50; i < sample_size + 50; i += 2) {
				println(i << ": " << reinterpret_cast<float*>(testMemoryHost)[i] << "|" << reinterpret_cast<float*>(testMemoryHost)[i + 1]);
			}
			println("");
			#if SHOW_DEBUG_OUTPUT == TRUE
			FILE* pFile;
			#if defined(__NVCC__)
			pFile = fopen("Result_Cuda.txt", "w");
			#else
			pFile = fopen("Result_Vulkan.txt", "w");
			#endif
			for (uint32_t i = 0; i < sample_size; ++i) {
				fprintf(pFile, "%i", (int)roundf(reinterpret_cast<float*>(testMemoryHost)[i] / normalisation_float + *correction_float_dev) & 1);
				if (i % 192 == 191) fprintf(pFile, "\n");
			}
			fclose(pFile);
			exit(0);
			abort();
			#endif
			assertTrue(isFletcherFloat(reinterpret_cast<float*>(testMemoryHost), sample_size, 8112419221.92300797, 20000.0, 542186359506315456.0, 2000000000000.0));
			assertTrue(isSha3(reinterpret_cast<uint8_t*>(key_rest[input_cache_read_pos_key]), vertical_len / 8, key_rest_hash));
			assertGPU(reinterpret_cast<uint32_t*>(correction_float_dev), 1, 0x3F54D912); //0.83143723	
		}		
		#endif
		if (do_xor_key_rest) {
			#if defined(__NVCC__)
			ToBinaryArray KERNEL_ARG4((int)((int)(vertical_block) / 31) + 1, 1023, 0, ToBinaryArrayStream)
				(invOut, reinterpret_cast<uint32_t*>(Output[output_cache_write_pos]), key_rest[input_cache_read_pos_key], correction_float_dev);
			#else
			vuda::launchKernel("SPIRV/toBinaryArray.spv", "main", ToBinaryArrayStream, (int)((int)(vertical_block) / 31) + 1, 1023, invOut, Output[output_cache_write_pos], key_rest[input_cache_read_pos_key], correction_float_dev, normalisation_float_dev);
			#endif
		} else {
			#if defined(__NVCC__)
				ToBinaryArrayNoXOR KERNEL_ARG4((int)((int)(do_compress ? vertical_block : desired_block / 2) / 31) + 1, 1023, 0, ToBinaryArrayStream)
					(invOut, reinterpret_cast<uint32_t*>(Output[output_cache_write_pos]), correction_float_dev);
			#else
			vuda::launchKernel("SPIRV/toBinaryArrayNoXOR.spv", "main", ToBinaryArrayStream, (int)((int)(do_compress ? vertical_block : desired_block / 2) / 31) + 1, 1023, invOut, Output[output_cache_write_pos], correction_float_dev, normalisation_float_dev);
			#endif
		}
		hipStreamSynchronize(ToBinaryArrayStream);
		#ifdef TEST
		if (doTest) {
			assertTrue(isSha3(reinterpret_cast<uint8_t*>(Output[output_cache_write_pos]), vertical_len / 8, ampout_sha3));
		}
		#endif
		//printBin(reinterpret_cast<uint8_t*>(Output[output_cache_write_pos]), reinterpret_cast<uint8_t*>(Output + output_cache_block_size * output_cache_write_pos) + 200);
		STOPWATCH_SAVE(stopwatch_toBinaryArray)
		STOPWATCH_TOTAL(stopwatch_total)

		
		#if STOPWATCH == TRUE
		if (stopwatch_total < stopwatch_total_max)
		{
			stopwatch_total_max = stopwatch_total;
			println(fixed << setprecision(3) <<
					"wait_for_input_buffer    " << stopwatch_wait_for_input_buffer / 1000000.0 << " ms\n" <<
					"cleaned_memory           " << stopwatch_cleaned_memory / 1000000.0 << " ms\n" <<
					"set_count_to_zero        " << stopwatch_set_count_one_to_zero / 1000000.0 << " ms\n" <<
				    "binIntffloat_seed        " << stopwatch_binInt2float_seed / 1000000.0 << " ms\n" <<
					"binIntffloat_key         " << stopwatch_binInt2float_key / 1000000.0 << " ms\n" <<
					"calculateCorrectionFloat " << stopwatch_calculateCorrectionFloat / 1000000.0 << " ms\n" <<
					"fft_key                  " << stopwatch_fft_key / 1000000.0 << " ms\n" <<
					"fft_seed                 " << stopwatch_fft_seed / 1000000.0 << " ms\n" <<
					"setFirstElementToZero    " << stopwatch_setFirstElementToZero / 1000000.0 << " ms\n" <<
					"elementWiseProduct       " << stopwatch_elementWiseProduct / 1000000.0 << " ms\n" <<
					"ifft                     " << stopwatch_ifft / 1000000.0 << " ms\n" <<
					"wait_for_output_buffer   " << stopwatch_wait_for_output_buffer / 1000000.0 << " ms\n" <<
					"toBinaryArray            " << stopwatch_toBinaryArray / 1000000.0 << " ms\n" <<
					"Total                    " << stopwatch_total / 1000000.0 << " ms\n" <<
					"Speed                    " << (1000000000.0 / stopwatch_total) * (sample_size / 1000000.0) << " MBit/s");
		}
		#endif


		if (speedtest) {
			speedtest_stop = chrono::high_resolution_clock::now();
			if (speedtest_nr >= 0) {
				uint32_t duration = chrono::duration_cast<chrono::microseconds>(speedtest_stop - speedtest_start).count();
				println("d[" << speedtest_i << "," << speedtest_j << "," << speedtest_nr << "]=" << (1000000.0 / duration) * (sample_size / 1000000.0));
			}
			if (speedtest_nr >= 9) {
				#if defined(__NVCC__)
				// Delete CUFFT Plans
				hipfftDestroy(plan_forward_R2C);
				hipfftDestroy(plan_inverse_C2R);
				#else
				// Delete CUFFT Plans
				deleteVkFFT(&plan_forward_R2C_seed);
				deleteVkFFT(&plan_forward_R2C_key);
				deleteVkFFT(&plan_inverse_C2R);
				#endif
				return;
			}
		}
		else
		{
			if (reuse_seed_amount > 0) {
				--reuse_seed_amount;
			}
			output_cache_write_pos = (output_cache_write_pos + 1) % output_blocks_to_cache;
		}

	}

	#if defined(__NVCC__)
	// Delete CUFFT Plans
	hipfftDestroy(plan_forward_R2C);
	hipfftDestroy(plan_inverse_C2R);
	#else
	// Delete CUFFT Plans
	deleteVkFFT(&plan_forward_R2C_seed);
	deleteVkFFT(&plan_forward_R2C_key);
	deleteVkFFT(&plan_inverse_C2R);
	#endif

}
